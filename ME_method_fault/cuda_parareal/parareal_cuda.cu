#include "hip/hip_runtime.h"
#include "basic_include.h"

void DifferentialEquations(float *delta_present,float *omega_present,float *Edp_present, float *Eqp_present, float delta_previous,float omega_previous,float Edp_previous, float Eqp_previous, float dt, float omega0, const float x_e)
{
	float I_q_predic, I_d_predic, d_delta_predic, d_omega_predic, d_Edp_predic, d_Eqp_predic, Te_predic;
	float I_q_correc, I_d_correc, d_delta_correc, d_omega_correc, d_Edp_correc, d_Eqp_correc;
	float Edp_predic, Eqp_predic, delta_predic, omega_predic, Te;

	/* Predictor Code*/
	I_d_predic=(Eb*cos(delta_previous) - Eqp_previous)/(x_dp+x_e);
	I_q_predic=(Edp_previous+Eb*sin(delta_previous))/(x_qp+x_e);

	d_Eqp_predic = 1/T_dp*(-Eqp_previous+ (x_d-x_dp)*I_d_predic+Efd);
	Eqp_predic = Eqp_previous + dt*d_Eqp_predic;

	d_Edp_predic = 1/T_qp*(-Edp_previous-(x_q-x_qp)*I_q_predic);
	Edp_predic = Edp_previous +dt*d_Edp_predic;

	Te_predic = Eqp_previous*I_q_predic+Edp_previous*I_d_predic+(x_dp-x_qp)*I_d_predic*I_q_predic;

	d_omega_predic= (1/(2*H))*(0.9- Te_predic);
	omega_predic= omega_previous + dt*d_omega_predic;

	d_delta_predic = 2*pi*60*(omega_previous - omega0);
	delta_predic = delta_previous + dt*d_delta_predic;

	/*Corrector Code*/

	I_d_correc=(Eb*cos(delta_predic) - Eqp_predic)/(x_dp+x_e);
	I_q_correc=(Edp_predic+Eb*sin(delta_predic))/(x_qp+x_e);


	d_Eqp_correc = 1/T_dp*(-Eqp_predic+ (x_d-x_dp)*I_d_correc+Efd);
	*Eqp_present = Eqp_previous + (dt/2)*(d_Eqp_predic+d_Eqp_correc);

	d_Edp_correc = 1/T_qp*(-Edp_predic-(x_q-x_qp)*I_q_correc);
	*Edp_present = Edp_previous +(dt/2)*(d_Edp_predic+d_Edp_correc);

	Te = Eqp_predic*I_q_correc+Edp_predic*I_d_correc+(x_dp-x_qp)*I_d_correc*I_q_correc;
	d_omega_correc= (1/(2*H))*(0.9- Te);
	*omega_present= omega_previous + (dt/2)*(d_omega_predic+d_omega_correc);

	d_delta_correc = 2*pi*60*(omega_predic - omega0);
	*delta_present = delta_previous + (dt/2)*(d_delta_correc+d_delta_predic);

}


__device__ void DifferentialEquationsGPU(float *delta_present,float *omega_present,float *Edp_present, float *Eqp_present, float delta_previous,float omega_previous,float Edp_previous, float Eqp_previous, float dt, float omega0, const float x_e)
{
	float I_q_predic, I_d_predic, d_delta_predic, d_omega_predic, d_Edp_predic, d_Eqp_predic, Te_predic;
	float I_q_correc, I_d_correc, d_delta_correc, d_omega_correc, d_Edp_correc, d_Eqp_correc;
	float Edp_predic, Eqp_predic, delta_predic, omega_predic, Te;

	/* Predictor Code*/
	I_d_predic=(Eb*cos(delta_previous) - Eqp_previous)/(x_dp+x_e);
	I_q_predic=(Edp_previous+Eb*sin(delta_previous))/(x_qp+x_e);

	d_Eqp_predic = 1/T_dp*(-Eqp_previous+ (x_d-x_dp)*I_d_predic+Efd);
	Eqp_predic = Eqp_previous + dt*d_Eqp_predic;

	d_Edp_predic = 1/T_qp*(-Edp_previous-(x_q-x_qp)*I_q_predic);
	Edp_predic = Edp_previous +dt*d_Edp_predic;

	Te_predic = Eqp_previous*I_q_predic+Edp_previous*I_d_predic+(x_dp-x_qp)*I_d_predic*I_q_predic;

	d_omega_predic= (1/(2*H))*(0.9- Te_predic);
	omega_predic= omega_previous + dt*d_omega_predic;

	d_delta_predic = 2*pi*60*(omega_previous - omega0);
	delta_predic = delta_previous + dt*d_delta_predic;

	/*Corrector Code*/

	I_d_correc=(Eb*cos(delta_predic) - Eqp_predic)/(x_dp+x_e);
	I_q_correc=(Edp_predic+Eb*sin(delta_predic))/(x_qp+x_e);


	d_Eqp_correc = 1/T_dp*(-Eqp_predic+ (x_d-x_dp)*I_d_correc+Efd);
	*Eqp_present = Eqp_previous + (dt/2)*(d_Eqp_predic+d_Eqp_correc);

	d_Edp_correc = 1/T_qp*(-Edp_predic-(x_q-x_qp)*I_q_correc);
	*Edp_present = Edp_previous +(dt/2)*(d_Edp_predic+d_Edp_correc);

	Te = Eqp_predic*I_q_correc+Edp_predic*I_d_correc+(x_dp-x_qp)*I_d_correc*I_q_correc;
	d_omega_correc= (1/(2*H))*(0.9- Te);
	*omega_present= omega_previous + (dt/2)*(d_omega_predic+d_omega_correc);

	d_delta_correc = 2*60*pi*(omega_predic - omega0);
	*delta_present = delta_previous + (dt/2)*(d_delta_correc+d_delta_predic);
}

__global__ void gpuparareal(float *g_delta,float *g_omega,float *g_Edp, float *g_Eqp, float *g_a,const float omega0,const float f_h,float *g_temp_delta,float *g_temp_omega,float *g_temp_Edp, float *g_temp_Eqp, float *g_delta_fine,float *g_omega_fine,float *g_Edp_fine, float *g_Eqp_fine,float *g_delta_diff,float *g_omega_diff,float *g_Edp_diff, float *g_Eqp_diff,int num_steps,const float c_h, const float x_edf, const float x_epf)
{
	const int idx = threadIdx.x + (blockIdx.x*blockDim.x);
	if(idx>=num_steps)
	{
		return;
	}
	float tempd,tempo,tempEdp,tempEqp,tint,tfin,fine_step,fine_tempd,fine_tempo, fine_tempEdp, fine_tempEqp;
	//      printf("g_a[%d] = %lf \n",idx,g_a[idx]);
	//      __syncthreads();
	tint = g_a[idx];
	tfin = g_a[idx+1];
	tempd = g_delta[idx];
	tempo = g_omega[idx];
	tempEdp = g_Edp[idx];
	tempEqp = g_Eqp[idx];
	bool flag = (g_a[idx]<=0.5);
	if(flag)
	{

		// printf("idx=%d  g_a = %lf g_a fin =%lf tint = %lf  tfinal = %lf \n\n",idx,g_a[idx],g_a[idx+1],tint,tfin);
		int umax = round((tfin-tint)/f_h);
		for (int u=0;u<umax;u++)
		{
			fine_step = tint+f_h;
			DifferentialEquationsGPU(&fine_tempd,&fine_tempo,&fine_tempEdp,&fine_tempEqp,tempd,tempo,tempEdp,tempEqp,f_h,omega0, x_edf);
			tempd=fine_tempd;
			tempo=fine_tempo;
			tempEdp=fine_tempEdp;
			tempEqp=fine_tempEqp;
			tint=fine_step;
			//printf("%f for time %f\n",(tempd*180)/pi,fine_step);
		}
		// printf("idx = %d The value of fine is %lf for time %lf\n",idx,(tempd*180/pi),tfin);
	}
	if(!flag)
	{

		// printf("idx=%d  g_a = %lf g_a fin =%lf tint = %lf  tfinal = %lf \n\n",idx,g_a[idx],g_a[idx+1],tint,tfin);
		int umax = round((tfin-tint)/f_h);
		for (int u=0;u<umax;u++)
		{
			fine_step = tint+f_h;
			DifferentialEquationsGPU(&fine_tempd,&fine_tempo,&fine_tempEdp,&fine_tempEqp,tempd,tempo,tempEdp,tempEqp,f_h,omega0, x_epf);
			tempd=fine_tempd;
			tempo=fine_tempo;
			tempEdp=fine_tempEdp;
			tempEqp=fine_tempEqp;
			tint=fine_step;
			//printf("%f for time %f\n",(tempd*180)/pi,fine_step);
		}
		// printf("idx = %d The value of fine is %lf for time %lf\n",idx,(tempd*180/pi),tfin);
	}

	g_delta_fine[idx+1]=tempd;
	g_omega_fine[idx+1]=tempo;
	g_Edp_fine[idx+1]=tempEdp;
	g_Eqp_fine[idx+1]=tempEqp;
	g_delta_diff[idx] = tempd - g_delta[idx+1];
	g_omega_diff[idx] = tempo - g_omega[idx+1];
	g_Edp_diff[idx] = tempEdp - g_Edp[idx+1];
	g_Eqp_diff[idx] = tempEqp - g_Eqp[idx+1];
	//      printf("idx = %d The value of fine is %lf for time %lf\n",idx,(tempd*180/pi),tfin);
	//      printf("The value of fine is %lf for time %lf\n",(tempd*180/pi),tfin);
}







__global__ void gpucorrection(float *d_delta_diff,float *d_omega_diff,float *d_Edp_diff, float *d_Eqp_diff,float *d_pred_delta,float *d_pred_omega,float *d_pred_Edp, float *d_pred_Eqp, float *d_correc_delta,float *d_correc_omega, float *d_correc_Edp, float *d_correc_Eqp, int num_steps)
{
	const int idx = threadIdx.x + (blockIdx.x*blockDim.x);
	if(idx>=num_steps)
	{
		return;
	}
	d_correc_delta[idx+1]=d_pred_delta[idx]+d_delta_diff[idx];
	d_correc_omega[idx+1]=d_pred_omega[idx]+d_omega_diff[idx];
	d_correc_Edp[idx+1] = d_Edp_diff[idx]+d_pred_Edp[idx];
	d_correc_Eqp[idx+1] = d_Eqp_diff[idx]+d_pred_Eqp[idx];
}




int main()

{
	hipEvent_t kernel_start;
	hipEvent_t kernel_stop;
	struct timeval start,stop;
	float tint, tfinal, faverage, fsum[10], et[110], cg_t, fg_t;
	float fElapsedTime;
	float fMemoryCopyTime[10];
	float fSequential_time[10];
	cout<<"Enter the coarse grid step size: "<< endl;
	cin>>cg_t;
	cout<<"Enter the fine grid step size: "<<endl;
	cin>>fg_t;
	cout<< "Enter the initial time step: "<< endl;
	cin>> tint;
	cout<<"Enter the final simulation time: "<<endl;
	cin>> tfinal;
	int coarse_steps = (tfinal-tint)/cg_t;
	int fine_steps = (tfinal-tint)/fg_t;
	size_t num_steps_bytes_coarse = coarse_steps*sizeof(float);
	size_t num_steps_bytes_fine = fine_steps*sizeof(float);

	// Initialization of variable for coarse funtion
	float *h_Edp = new float[coarse_steps];
	float *h_Eqp= new float[coarse_steps];
	float *h_omega = new float[coarse_steps];
	float *h_delta = new float[coarse_steps];
	float *d_Edp,*d_Eqp,*d_omega,*d_delta; 

	// Initialization of variables for fine funtion

	float *h_fine_Edp = new float[coarse_steps];
	float *h_fine_Eqp = new float[coarse_steps];
	float *h_fine_omega = new float[coarse_steps];
	float *h_fine_delta = new float[coarse_steps];
	float *h_temp_Edp = new float[fine_steps];
	float *h_temp_Eqp = new float[fine_steps];
	float *h_temp_delta = new float[fine_steps];
	float *h_temp_omega = new float[fine_steps];
	float *h_Edp_fine = new float[coarse_steps];
	float *h_Eqp_fine = new float[coarse_steps];
	float *h_omega_fine = new float[coarse_steps];
	float *h_delta_fine = new float[coarse_steps];

	float *d_fine_Edp,*d_fine_Eqp,*d_fine_omega,*d_fine_delta;
	float *d_temp_Edp,*d_temp_Eqp,*d_temp_omega,*d_temp_delta;
	float *d_Edp_fine,*d_Eqp_fine,*d_omega_fine,*d_delta_fine;



	// Predictor-Corrector initialization
	float *h_pred_Edp = new float[coarse_steps];
	float *h_pred_Eqp = new float[coarse_steps];
	float *h_pred_delta = new float[coarse_steps];
	float *h_pred_omega = new float[coarse_steps];
	float *h_correc_Edp = new float[coarse_steps];
	float *h_correc_Eqp = new float[coarse_steps];
	float *h_correc_delta = new float[coarse_steps];
	float *h_correc_omega = new float[coarse_steps];
	float *h_Edp_diff = new float[coarse_steps];
	float *h_Eqp_diff = new float[coarse_steps];
	float *h_delta_diff = new float[coarse_steps];
	float *h_omega_diff = new float[coarse_steps];


	float *d_pred_Edp,*d_pred_Eqp,*d_pred_omega,*d_pred_delta;
	float *d_correc_Edp,*d_correc_Eqp,*d_correc_omega,*d_correc_delta;
	float *d_Edp_diff,*d_Eqp_diff,*d_omega_diff,*d_delta_diff;
	float *d_a;
	float x_edf, x_epf;
	x_edf = 0.9;
	x_epf = 0.5;
	float omega0=0;
	h_omega[0]=omega0;
	h_delta[0]=(56.1210*pi)/180;
	h_Edp[0]= -0.4973;
	h_Eqp[0]= 1.0337;
	int kmax=2;
	float *h_a = new float[coarse_steps];
	h_a[0]=tint;
	
	for (int i=0;i<coarse_steps;i++)
	{
	h_a[i+1] = h_a[i]+cg_t;
	}
		
	for (int k=0;k<kmax;k++)
	{	
		// Coarse Solver
		gettimeofday(&start,NULL);
		if(k==0)
		{	
			gettimeofday(&start,NULL);
			for (int i=0;i<coarse_steps;i++)
			{
				//h_a[i+1] = h_a[i]+cg_t;
				if(h_a[i+1]<=0.5)
				{
				DifferentialEquations(&h_delta[i+1],&h_omega[i+1], &h_Edp[i+1], &h_Eqp[i+1],h_delta[i],h_omega[i],h_Edp[i], h_Eqp[i], cg_t, omega0, x_edf);
				//    cout<< "The coarse grid values are "<< (h_delta[i+1]*180)/pi<<endl;
				}
				if(h_a[i+1]>0.5)
                                {
                                DifferentialEquations(&h_delta[i+1],&h_omega[i+1], &h_Edp[i+1], &h_Eqp[i+1],h_delta[i],h_omega[i],h_Edp[i], h_Eqp[i], cg_t, omega0,x_epf);
                                //    cout<< "The coarse grid values are "<< (h_delta[i+1]*180)/pi<<endl;
                                }

			}
				gettimeofday(&stop,NULL);
			//	fSequential_time[k] = ((end.tv_sec*1e6+end.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		}

		else
		{
			gettimeofday(&start,NULL);
			for (int i=k;i<coarse_steps;i++)
			{
				h_Edp[i] = h_correc_Edp[i];
				h_Eqp[i] = h_correc_Eqp[i];
				h_delta[i] = h_correc_delta[i];
				h_omega[i] = h_correc_omega[i];
				//h_a[i+1]=h_a[i]+cg_t;
				if(h_a[i+1]<=0.5)
                                {
				DifferentialEquations(&h_delta[i+1],&h_omega[i+1], &h_Edp[i+1], &h_Eqp[i+1],h_delta[i],h_omega[i],h_Edp[i], h_Eqp[i], cg_t, omega0, x_edf);
				// cout<< "The coarse grid values for else case are "<< (delta[i+1]*180)/pi<<endl;
				}
				if(h_a[i+1]>0.5)
                                {
                                DifferentialEquations(&h_delta[i+1],&h_omega[i+1], &h_Edp[i+1], &h_Eqp[i+1],h_delta[i],h_omega[i],h_Edp[i], h_Eqp[i], cg_t, omega0,x_epf);
                                //    cout<< "The coarse grid values are "<< (h_delta[i+1]*180)/pi<<endl;
                                }

			}
			 gettimeofday(&stop,NULL);

		}
		gettimeofday(&stop,NULL);
		fSequential_time[k] = ((stop.tv_sec*1e6+stop.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		// Fine Solver
		CHECK(hipEventCreate(&kernel_start));
		CHECK(hipEventCreate(&kernel_stop));
		//Allocating memory on GPU for device variables

		CHECK(hipMalloc((float**)&d_delta,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_omega,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_Eqp,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_Edp,num_steps_bytes_coarse+4));

		CHECK(hipMalloc((float**)&d_temp_delta,num_steps_bytes_fine+4));
		CHECK(hipMalloc((float**)&d_temp_omega,num_steps_bytes_fine+4));
		CHECK(hipMalloc((float**)&d_temp_Edp,num_steps_bytes_fine+4));
		CHECK(hipMalloc((float**)&d_fine_Eqp,num_steps_bytes_fine+4));

		CHECK(hipMalloc((float**)&d_fine_delta,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_fine_omega,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_fine_Eqp,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_fine_Edp,num_steps_bytes_coarse+4));		

		CHECK(hipMalloc((float**)&d_delta_fine,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_omega_fine,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_Eqp_fine,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_Edp_fine,num_steps_bytes_coarse+4));

		CHECK(hipMalloc((float**)&d_a,num_steps_bytes_coarse+4));

		CHECK(hipMalloc((float**)&d_delta_diff,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_omega_diff,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_Edp_diff,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_Eqp_diff,num_steps_bytes_coarse+4));

		CHECK(hipMalloc((float**)&d_pred_delta,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_pred_omega,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_pred_Edp,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_pred_Eqp,num_steps_bytes_coarse+4));

		CHECK(hipMalloc((float**)&d_correc_delta,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_correc_omega,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_correc_Edp,num_steps_bytes_coarse+4));
		CHECK(hipMalloc((float**)&d_correc_Eqp,num_steps_bytes_coarse+4));
		//copying the data to device from host
		gettimeofday(&start,NULL);
		CHECK(hipMemcpy(d_delta,h_delta,num_steps_bytes_coarse+4,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_omega,h_omega,num_steps_bytes_coarse+4,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_Edp,h_Edp,num_steps_bytes_coarse+4,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_Eqp,h_Eqp,num_steps_bytes_coarse+4,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_a,h_a,num_steps_bytes_coarse+4,hipMemcpyHostToDevice));
		gettimeofday(&stop,NULL);
		fMemoryCopyTime[k] = ((stop.tv_sec*1e6+stop.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;


		int ilen = 256;
		dim3 block (ilen,1,1);
		dim3 grid ((coarse_steps+block.x-1)/block.x,1,1);
		/*		cout << "1D Grid Dimension" << endl;
				cout << "\tNumber of Blocks along X dimension: " << grid.x << endl;
				cout << "1D Block Dimension" << endl;
				cout << "\tNumber of threads along X dimension: " << block.x << endl;
		 */
		et[0]=0;
		for(int i=0;i<110;i++)
		{	
			et[i]=0;
			CHECK(hipEventRecord(kernel_start));
			gpuparareal<<<grid,block>>>(d_delta,d_omega,d_Edp,d_Eqp,d_a,omega0,fg_t,d_temp_delta,d_temp_omega,d_temp_Edp,d_temp_Eqp,d_delta_fine,d_omega_fine,d_Edp_fine, d_Eqp_fine, d_delta_diff,d_omega_diff,d_Edp_diff,d_Eqp_diff, coarse_steps,cg_t, x_edf,x_epf);
			CHECK(hipEventRecord(kernel_stop));
			CHECK(hipEventSynchronize(kernel_stop));
			CHECK(hipEventElapsedTime(&fElapsedTime,kernel_start,kernel_stop));
			et[i]=fElapsedTime;
			//		cout<<"Et is "<< et<<endl;
		}


		/*CHECK(hipMemcpy(h_temp_delta,d_temp_delta,num_steps_bytes_fine+4,hipMemcpyDeviceToHost));
		  CHECK(hipMemcpy(h_temp_omega,d_temp_omega,num_steps_bytes_fine+4,hipMemcpyDeviceToHost));
		  CHECK(hipMemcpy(h_temp_Edp,d_temp_Edp,num_steps_bytes_fine+4,hipMemcpyDeviceToHost));
		  CHECK(hipMemcpy(h_temp_Eqp,d_temp_Eqp,num_steps_bytes_fine+4,hipMemcpyDeviceToHost));
		 */
		gettimeofday(&start,NULL);
		CHECK(hipMemcpy(h_delta_diff,d_delta_diff,num_steps_bytes_coarse+4,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_omega_diff,d_omega_diff,num_steps_bytes_coarse+4,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_Eqp_diff,d_Eqp_diff,num_steps_bytes_coarse+4,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_Edp_diff,d_Edp_diff,num_steps_bytes_coarse+4,hipMemcpyDeviceToHost));

		CHECK(hipMemcpy(h_delta_fine,d_delta_fine,num_steps_bytes_coarse+4,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_omega_fine,d_omega_fine,num_steps_bytes_coarse+4,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_Eqp_fine,d_Eqp_fine,num_steps_bytes_coarse+4,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_Edp_fine,d_Edp_fine,num_steps_bytes_coarse+4,hipMemcpyDeviceToHost));
		gettimeofday(&stop,NULL);

		fMemoryCopyTime[k]+= ((stop.tv_sec*1e6+stop.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		//                cout<< "Memory transfer time =  " << fMemoryCopyTime[k] <<" ms"<<endl;		
		h_pred_delta[k]=h_delta[k];
		h_pred_omega[k]=h_omega[k];
		h_pred_Edp[k]=h_Edp[k];
		h_pred_Eqp[k]=h_Eqp[k];
		gettimeofday(&start,NULL);
		for (int i=k;i<coarse_steps;i++)
		{
			if(h_a[i+1]<=0.5)
                        {
                         
			//      Differential(&pred_delt[i+1],&pred_omega[i+1],pred_delt[i],pred_omega[i],omega0,c_h);
			DifferentialEquations(&h_pred_delta[i+1],&h_pred_omega[i+1], &h_pred_Edp[i+1], &h_pred_Eqp[i+1],h_pred_delta[i],h_pred_omega[i],h_pred_Edp[i], h_pred_Eqp[i], cg_t, omega0, x_edf);
			}
			
			if(h_a[i+1]>0.5)
                        {

                        //      Differential(&pred_delt[i+1],&pred_omega[i+1],pred_delt[i],pred_omega[i],omega0,c_h);
                        DifferentialEquations(&h_pred_delta[i+1],&h_pred_omega[i+1], &h_pred_Edp[i+1], &h_pred_Eqp[i+1],h_pred_delta[i],h_pred_omega[i],h_pred_Edp[i], h_pred_Eqp[i], cg_t, omega0, x_epf);
                        }
		}
		gettimeofday(&stop,NULL);
		fSequential_time[k] += ((stop.tv_sec*1e6+stop.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;

		/*	for (int i=0;i<coarse_steps;i++)
			{
			correc_delta[i+1] = delta_diff[i]+pred_delta[i];
			correc_omega[i+1] = omega_diff[i]+pred_omega[i];
			correc_Edp[i+1] = Edp_diff[i]+pred_Edp[i];
			correc_Eqp[i+1] = Eqp_diff[i]+pred_Eqp[i];
		//cout<< "The corrected grid values are "<< (corec_delt[i+1]*180)/pi<<" for time"<<a[i+1]<<"for array element "<<i<<endl;
		}
		 */

		gettimeofday(&start,NULL);
		CHECK(hipMemcpy(d_delta_diff,h_delta_diff,num_steps_bytes_coarse+4,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_omega_diff,h_omega_diff,num_steps_bytes_coarse+4,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_Edp_diff,h_Edp_diff,num_steps_bytes_coarse+4,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_Eqp_diff,h_Eqp_diff,num_steps_bytes_coarse+4,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_pred_delta,h_pred_delta,num_steps_bytes_coarse+4,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_pred_omega,h_pred_omega,num_steps_bytes_coarse+4,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_pred_Edp,h_pred_Edp,num_steps_bytes_coarse+4,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_pred_Eqp,h_pred_Eqp,num_steps_bytes_coarse+4,hipMemcpyHostToDevice));
		gettimeofday(&stop,NULL);
		float time = 0;
		time = ((stop.tv_sec*1e6+stop.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		//      int ilen = 256;
		//      dim3 block (ilen,1,1);
		//    dim3 grid ((num_steps+block.x-1)/block.x,1,1);
		CHECK(hipEventRecord(kernel_start));
		gpucorrection<<<grid,block>>>(d_delta_diff,d_omega_diff,d_Edp_diff, d_Eqp_diff, d_pred_delta,d_pred_omega,d_Edp,d_Eqp, d_correc_delta,d_correc_omega,d_correc_Edp,d_correc_Eqp,coarse_steps);
		CHECK(hipEventRecord(kernel_stop));
		CHECK(hipEventSynchronize(kernel_stop));
		CHECK(hipEventElapsedTime(&fElapsedTime,kernel_start,kernel_stop));
		//	cout<<"Elapsed time is for correction is " <<fElapsedTime<<" ms"<<endl;
		gettimeofday(&start,NULL);
		CHECK(hipMemcpy(h_correc_delta,d_correc_delta,num_steps_bytes_coarse+4,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_correc_omega,d_correc_omega,num_steps_bytes_coarse+4,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_correc_Edp,d_correc_Edp,num_steps_bytes_coarse+4,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_correc_Eqp,d_correc_Eqp,num_steps_bytes_coarse+4,hipMemcpyDeviceToHost));
		gettimeofday(&stop,NULL);
		time+= ((stop.tv_sec*1e6+stop.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		// cout<<"Correction memory copy time is: "<<time<<" ms"<<endl;

		for(int i=k;i<coarse_steps;i++)
		{
			if(abs(h_delta[i+1]-h_correc_delta[i+1])<1e-6)
			{
				if(i<1000)
				{
				        cout<<"converged for "<<i+1<<"\telement"<<" for iteration: "<<k<<endl;
					break;
				}
			}
		}

		//		fSequential_time[k] += ((stop.tv_sec*1e6+stop.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		faverage = 0;	
		for(int i=10;i<110;i++)
		{
			faverage+=et[i];
		}
		fsum[k]=faverage/100;
		cout<<"The gpu execution time  for fine grid is: "<<fsum[k]<<" ms"<<"\t"<<"sequential time for coarse + predictor is: "<<fSequential_time[k]<<" ms"<<endl;
		cout<<"Memory transfer time for fine grid:  " << fMemoryCopyTime[k] <<" ms"<<"\t"<<"Memory transfer for Corrector is: "<<time<<" ms"<<endl;
		cout<<"Elapsed time is for correction on GPU is: " <<fElapsedTime<<" ms"<<endl;
		//cout<<"Correction memory copy time is: "<<time<<" ms"<<endl;
	}
/*		for (int i=0;i<coarse_steps;i++)
		{
		cout<< "Delta values are "<< (h_delta[i]*180)/pi<<endl;
		}
*/

	CHECK(hipEventDestroy(kernel_start));
	CHECK(hipEventDestroy(kernel_stop));
	CHECK(hipFree(d_omega));
	CHECK(hipFree(d_delta));
	CHECK(hipFree(d_Edp));
	CHECK(hipFree(d_Eqp));
	CHECK(hipFree(d_temp_delta));
	CHECK(hipFree(d_temp_omega));
	//CHECK(hipFree(d_temp_Edp));
	//CHECK(hipFree(d_temp_Eqp));
	CHECK(hipFree(d_delta_fine));
	CHECK(hipFree(d_omega_fine));
	CHECK(hipFree(d_Edp_fine));
	CHECK(hipFree(d_Eqp_fine));
	CHECK(hipFree(d_a));
	CHECK(hipFree(d_delta_diff));
	CHECK(hipFree(d_omega_diff));
	CHECK(hipFree(d_Edp_diff));
	CHECK(hipFree(d_Eqp_diff));
	CHECK(hipFree(d_fine_delta));
	CHECK(hipFree(d_fine_omega));
	CHECK(hipFree(d_fine_Edp));
	CHECK(hipFree(d_fine_Eqp));
	CHECK(hipFree(d_pred_delta));
	CHECK(hipFree(d_pred_omega));
	CHECK(hipFree(d_pred_Edp));
	CHECK(hipFree(d_pred_Eqp));
	CHECK(hipFree(d_correc_delta));
	CHECK(hipFree(d_correc_omega));
	CHECK(hipFree(d_correc_Edp));
	CHECK(hipFree(d_correc_Eqp));
	CHECK(hipDeviceReset());

	delete[] h_Eqp;
	delete[] h_Edp;
	delete[] h_omega;
	delete[] h_delta;
	delete[] h_Eqp_fine;
	delete[] h_Edp_fine;
	delete[] h_omega_fine;
	delete[] h_delta_fine;
	delete[] h_temp_Eqp;
	delete[] h_temp_Edp;
	delete[] h_temp_omega;
	delete[] h_temp_delta;
	delete[] h_fine_Eqp;
	delete[] h_fine_Edp;
	delete[] h_fine_omega;
	delete[] h_fine_delta;
	delete[] h_Eqp_diff;
	delete[] h_Edp_diff;
	delete[] h_omega_diff;
	delete[] h_delta_diff;
	delete[] h_pred_Eqp;
	delete[] h_pred_Edp;
	delete[] h_pred_omega;
	delete[] h_pred_delta;
	delete[] h_correc_Eqp;
	delete[] h_correc_Edp;
	delete[] h_correc_omega;
	delete[] h_correc_delta;
	delete[] h_a;
}
