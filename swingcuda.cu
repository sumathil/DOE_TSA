#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "iostream"
#include "iomanip"
#include "cmath"
#include <stdio.h>
using namespace std;
#define pi 3.14159265358979323846
#define CHECK(call) \
{                                                                        \
	const hipError_t error = call;                                       \
	if (error != hipSuccess)                                             \
	{                                                                     \
		printf("Error: %s:%d, ", __FILE__, __LINE__);                      \
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
		exit(1);                                                           \
	}                                                                     \
}


// When the fault is occured for GPU function
__device__ void Differentiald(double *deltapresent,double *omegapresent,double deltaprevious,double omegaprevious,double omega0,double c_h)
{
	double temp,temp1,ddeltapresent,domegapresent,ddeltaprevious,domegaprevious;
	ddeltaprevious =omegaprevious-omega0;
	temp=deltaprevious+(c_h)*(ddeltaprevious);
	domegaprevious =((pi*60)/5)* (0.8-0.65*sin(temp));
	temp1 = omegaprevious+(c_h*(domegaprevious));
	ddeltapresent = temp1-omegaprevious;
	*deltapresent = deltaprevious + (c_h/2)*(ddeltaprevious+ddeltapresent);
	domegapresent =((pi*60)/5)* (0.8-(0.65*sin(*deltapresent)));
	//domegapresent = 32-(173.68*sin(*deltapresent-(10*pi)/180));
	*omegapresent = omegaprevious+(c_h/2)*(domegaprevious+domegapresent);
}



//Once the fault is cleared for GPU function

__device__ void Differentialpostfaultd(double *deltapresent,double *omegapresent,double deltaprevious,double omegaprevious,double omega0,double c_h)
{
	double temp,temp1,ddeltapresent,domegapresent,ddeltaprevious,domegaprevious;
	ddeltaprevious =omegaprevious-omega0;
	temp=deltaprevious+(c_h)*(ddeltaprevious);
	domegaprevious =((pi*60)/5)* (0.8-1.4625*sin(temp));
	temp1 = omegaprevious+(c_h*(domegaprevious));
	ddeltapresent = temp1-omegaprevious;
	*deltapresent = deltaprevious + (c_h/2)*(ddeltaprevious+ddeltapresent);
	domegapresent =((pi*60)/5)* (0.8-(1.4625*sin(*deltapresent)));
	*omegapresent = omegaprevious+(c_h/2)*(domegaprevious+domegapresent);
}



__global__ void gpuparareal(double *g_delta,double *g_omega,double *g_a,const double omega0,const double f_h,double *g_fine_tempd,double *g_fine_tempo,double *g_del_fine,double *g_omega_fine,double *g_diff_delta,double *g_diff_omega,int num_steps,int c_h)
{
	const int idx = threadIdx.x + (blockIdx.x*blockDim.x);
	if(idx>=num_steps)
	{
		return;
	}
	double tempd,tempo,tint,tfin,fine_step,fine_tempd,fine_tempo;
	tint = g_a[idx];
	tfin = g_a[idx+1];
	tempd = g_delta[idx];
	tempo = g_omega[idx];
	//printf("g_a[%d] = %lf \n",idx,g_a[idx]);
	//	__syncthreads();
	bool flag = (g_a[idx]<0.8);
	if(flag)
	{
		int umax = round((tfin-tint)/f_h);
		for (int u=0;u<umax;u++)
		{
			fine_step = tint+f_h;
			Differentiald(&fine_tempd,&fine_tempo,tempd,tempo,omega0,f_h);
			tempd=fine_tempd;
			tempo=fine_tempo;
			tint=fine_step;
		}
		//   printf("idx = %d The value of fine is %lf for time %lf\n",idx,(tempd*180/pi),tfin);
	}

	if(!flag)
	{
		int umax = round((tfin-tint)/f_h);
		for (int u=0;u<umax;u++)
		{
			fine_step = tint+f_h;
			Differentialpostfaultd(&fine_tempd,&fine_tempo,tempd,tempo,omega0,f_h);
			tempd=fine_tempd;
			tempo=fine_tempo;
			tint=fine_step;
		}
	}
	g_del_fine[idx+1]=tempd;
	g_omega_fine[idx+1]=tempo;
	g_diff_delta[idx] = tempd - g_delta[idx+1];
	g_diff_omega[idx] = tempo - g_omega[idx+1];
	//	printf("idx = %d The value of fine is %lf for time %lf\n",idx,(tempd*180/pi),tfin);
	//printf("The value of fine is %lf for time %lf\n",(tempd*180/pi),tfin);

}

__global__ void gpucorrection(double *d_diff_delta,double *d_diff_omega,double *d_pred_delta,double *d_pred_omega,double *d_corec_delta,double *d_corec_omega, int num_steps)
{
	const int idx = threadIdx.x + (blockIdx.x*blockDim.x);
	if(idx>=num_steps)
	{
		return;
	}
	d_corec_delta[idx+1]=d_pred_delta[idx]+d_diff_delta[idx];
	d_corec_omega[idx+1]=d_pred_omega[idx]+d_diff_omega[idx];
}



// When the fault is occured
void Differential(double *deltapresent,double *omegapresent,double deltaprevious,double omegaprevious,double omega0,double c_h)
{
	double temp,temp1,ddeltapresent,domegapresent,ddeltaprevious,domegaprevious;
	ddeltaprevious =omegaprevious-omega0;
	temp=deltaprevious+(c_h)*(ddeltaprevious);
	domegaprevious =((pi*60)/5)* (0.8-0.65*sin(temp));
	temp1 = omegaprevious+(c_h*(domegaprevious));
	ddeltapresent = temp1-omegaprevious;
	*deltapresent = deltaprevious + (c_h/2)*(ddeltaprevious+ddeltapresent);
	domegapresent =((pi*60)/5)* (0.8-(0.65*sin(*deltapresent)));
	//domegapresent = 32-(173.68*sin(*deltapresent-(10*pi)/180));
	*omegapresent = omegaprevious+(c_h/2)*(domegaprevious+domegapresent);
}

//Once the fault is cleared

void Differentialpostfault(double *deltapresent,double *omegapresent,double deltaprevious,double omegaprevious,double omega0,double c_h)
{
	double temp,temp1,ddeltapresent,domegapresent,ddeltaprevious,domegaprevious;
	ddeltaprevious =omegaprevious-omega0;
	temp=deltaprevious+(c_h)*(ddeltaprevious);
	domegaprevious =((pi*60)/5)* (0.8-1.4625*sin(temp));
	temp1 = omegaprevious+(c_h*(domegaprevious));
	ddeltapresent = temp1-omegaprevious;
	*deltapresent = deltaprevious + (c_h/2)*(ddeltaprevious+ddeltapresent);
	domegapresent =((pi*60)/5)* (0.8-(1.4625*sin(*deltapresent)));
	*omegapresent = omegaprevious+(c_h/2)*(domegaprevious+domegapresent);
}



int main()
{
	hipEvent_t kernel_start;
	hipEvent_t kernel_stop;
	struct timeval start,end;
	double tint,tfin,omega0;
	float fElapsedTime,faverage,fsum[10];
	double fMemoryCopyTime[10];
	double fSequential_time[10],tet[10];
	//host variables
	double *h_omega,*h_delta,*h_a,c_h,f_h,dint,*h_del_fine,*h_omega_fine,*h_diff_delta,*h_diff_omega;
	//device variables
	double *d_omega,*d_delta,*d_a,*d_del_fine,*d_omega_fine,*d_fine_tempd,*d_fine_tempo,*d_diff_delta,*d_diff_omega,*d_pred_delta,*d_corec_delta,*d_pred_omega,*d_corec_omega;
	double *h_pred_delta,*h_corec_delta,*h_pred_omega,*h_corec_omega,*h_fine_tempd,*h_fine_tempo,et[110];
	cout<<"The initial time value is : "<<endl;
	cin>>tint;
	cout<<"The final time value is: "<<endl;
	cin>>tfin;
	cout<<"The coarse grid time step value is: "<<endl;
	cin>>c_h;
	cout<<"The fine grid step size value is: "<<endl;
	cin>>f_h;
	cout<<"Enter the intial value of delta in degrees: "<<endl;
	cin>>dint;
	int num_steps = ((tfin-tint)/c_h)+1;
	cout<<"the number of steps for coarse : "<<num_steps<<endl;
	size_t num_steps_bytes_coarse = num_steps*sizeof(double);
	int fine_size = ((tfin-tint)/f_h)+1;
	cout<<"The number of steps for fine : "<<fine_size<<endl;
	size_t num_steps_bytes_fine = fine_size*sizeof(double);
	h_omega = new double[num_steps];
	h_delta = new double[num_steps];
	h_a = new double[num_steps];
	h_del_fine = new double[num_steps];
	h_omega_fine = new double[num_steps];
	h_fine_tempd= new double[fine_size];
	h_fine_tempo=new double[fine_size];
	h_pred_delta= new double[num_steps];
	h_pred_omega= new double[num_steps];
	h_corec_delta=new double[num_steps];
	h_corec_omega = new double[num_steps];
	h_diff_delta = new double[num_steps];
	h_diff_omega = new double [num_steps];
	omega0=2*pi*60;
	h_omega[0]=omega0;
	h_delta[0]=(dint*pi)/180;
	cout<<"The value in radians is: "<<h_delta[0]<<endl;
	h_a[0] =0;
	h_a[0]=tint;
	num_steps =num_steps - 1;
	fine_size =fine_size - 1;
	cout<<num_steps<<endl;	
	for(int k=0;k<2;k++)
	{
		fMemoryCopyTime[k]=0;	
		fSequential_time[k]=0;
		//	gettimeofday(&start,NULL);
		if(k==0)
		{ 	
			gettimeofday(&start,NULL);
			for (int i=0;i<num_steps;i++)
			{
				h_a[i+1]=h_a[i]+c_h; //a[i] contains all the time step required for coarse grid calculation
				if(h_a[i+1]<=0.8)
				{
					//cout << "a= " <<h_a[i+1]<<__LINE__<<endl;
					//	h_a[i+1]=h_a[i]+c_h; //a[i] contains all the time step required for coarse grid calculation
					Differential(&h_delta[i+1],&h_omega[i+1],h_delta[i],h_omega[i],omega0,c_h);
					//cout<< "The coarse grid values are "<< (h_delta[i+1]*180)/pi<<" for time"<<h_a[i+1]<<"for array element "<<i<<"for k value "<<k<<endl;
					//cout<<"break 2"<<endl;

				}
				if(h_a[i+1]>0.8)
				{
					//cout << "a= " <<h_a[i]<<__LINE__<<endl;
					//h_a[i+1]=h_a[i]+c_h;
					Differentialpostfault(&h_delta[i+1],&h_omega[i+1],h_delta[i],h_omega[i],omega0,c_h);
					//cout<< "The coarse grid values are "<< (h_delta[i+1]*180)/pi<<" for time"<<h_a[i+1]<<"for array element "<<i<<"for k value "<<k<<endl; 
				}

			}
			gettimeofday(&end,NULL);
			fSequential_time[k] = ((end.tv_sec*1e6+end.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
			cout<<" The Sequential Execution time is : "<<fSequential_time[k]<<" ms"<<endl;
		}
		else
		{
			gettimeofday(&start,NULL);
			for(int i=1;i<num_steps;i++)
			{
				h_delta[i]=h_corec_delta[i];
				h_omega[i]=h_corec_omega[i];
				h_a[i+1]=h_a[i]+c_h; //a[i] contains all the time step required for coarse grid calculation
				if(h_a[i+1]<=0.8)
				{
					//	h_a[i+1]=h_a[i]+c_h; //a[i] contains all the time step required for coarse grid calculation
					Differential(&h_delta[i+1],&h_omega[i+1],h_delta[i],h_omega[i],omega0,c_h);
					//cout<< "The coarse grid values are "<< (h_delta[i+1]*180)/pi<<" for time"<<h_a[i+1]<<"for array element "<<i<<"for k value "<<k<<endl;

				}
				if(h_a[i+1]>0.8)
				{
					//h_a[i+1]=h_a[i]+c_h;
					Differentialpostfault(&h_delta[i+1],&h_omega[i+1],h_delta[i],h_omega[i],omega0,c_h);
					//cout<< "The coarse grid values are "<< (h_delta[i+1]*180)/pi<<" for time"<<h_a[i+1]<<"for array element "<<i<<"for k value "<<k<<endl; 
				}

			}
			gettimeofday(&end,NULL);
			fSequential_time[k] = ((end.tv_sec*1e6+end.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
			cout<<" The Sequential Execution time is : "<<fSequential_time[k]<<" ms"<<endl;
		}

		//cout<<" The Sequential Execution time is : "<<fSequential_time<<" ms"<<endl;
		//      cudaEvent_t kernel_start;
		//      cudaEvent_t kernel_stop;

		hipError_t hipSetDevice(int device);
		hipSetDevice(0);
		CHECK(hipEventCreate(&kernel_start));
		CHECK(hipEventCreate(&kernel_stop));
		//Allocating memory on GPU for device variables

		CHECK(hipMalloc((double**)&d_delta,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_omega,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_fine_tempd,num_steps_bytes_fine+8));
		CHECK(hipMalloc((double**)&d_fine_tempo,num_steps_bytes_fine+8));
		CHECK(hipMalloc((double**)&d_del_fine,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_omega_fine,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_a,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_diff_delta,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_diff_omega,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_pred_delta,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_pred_omega,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_corec_delta,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_corec_omega,num_steps_bytes_coarse+8));
		//copying the data to device from host
		gettimeofday(&start,NULL);
		CHECK(hipMemcpy(d_delta,h_delta,num_steps_bytes_coarse+8,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_omega,h_omega,num_steps_bytes_coarse+8,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_a,h_a,num_steps_bytes_coarse+8,hipMemcpyHostToDevice));
		gettimeofday(&end,NULL);
		fMemoryCopyTime[k] = ((end.tv_sec*1e6+end.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		//Kernel call
		int ilen = 256;
		dim3 block (ilen,1,1);
		dim3 grid ((num_steps+block.x-1)/block.x,1,1);
		cout << "1D Grid Dimension" << endl;
		cout << "\tNumber of Blocks along X dimension: " << grid.x << endl;
		cout << "1D Block Dimension" << endl;
		cout << "\tNumber of threads along X dimension: " << block.x << endl;
		//kernel function
		et[0]=0;
		for(int i=0;i<110;i++)
		{
			CHECK(hipEventRecord(kernel_start));
			gpuparareal<<<grid,block>>>(d_delta,d_omega,d_a,omega0,f_h,d_fine_tempd,d_fine_tempo,d_del_fine,d_omega_fine,d_diff_delta,d_diff_omega,num_steps,c_h);
			CHECK(hipEventRecord(kernel_stop));
			CHECK(hipEventSynchronize(kernel_stop));
			CHECK(hipEventElapsedTime(&fElapsedTime,kernel_start,kernel_stop));
			et[i]=fElapsedTime;	
		}
		//	cout << "Kernel with Compiler Implementation = " << fElapsedTime << " msecs" << endl;
		//	gettimeofday(&start,NULL);
		CHECK(hipMemcpy(h_fine_tempd,d_fine_tempd,num_steps_bytes_fine+8,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_fine_tempo,d_fine_tempo,num_steps_bytes_fine+8,hipMemcpyDeviceToHost));
		gettimeofday(&start,NULL);
		CHECK(hipMemcpy(h_diff_delta,d_diff_delta,num_steps_bytes_coarse+8,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_diff_omega,d_diff_omega,num_steps_bytes_coarse+8,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_del_fine,d_del_fine,num_steps_bytes_coarse+8,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_omega_fine,d_omega_fine,num_steps_bytes_coarse+8,hipMemcpyDeviceToHost));
		gettimeofday(&end,NULL);
		fMemoryCopyTime[k]+= ((end.tv_sec*1e6+end.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		cout<< "Memory transfer time =  " << fMemoryCopyTime[k] <<" ms"<<endl;
		h_pred_delta[0]=h_del_fine[1];
		h_pred_omega[0]=h_omega_fine[1];
		cout<<"Fine values are: "<<"\tdelta"<< (h_del_fine[1]*180/pi)<<"\t omega"<<h_omega_fine[1]<<endl;
		gettimeofday(&start,NULL);
		for (int i=1;i<num_steps;i++)
		{
			h_a[i+1]=h_a[i]+c_h; //a[i] contains all the time step required for coarse grid calculation
			if(h_a[i+1]<=0.8)
			{
				//h_a[i+1]=h_a[i]+c_h;
				Differential(&h_pred_delta[i],&h_pred_omega[i],h_pred_delta[i-1],h_pred_omega[i-1],omega0,c_h);
				//cout<<"The predicted value is "<<(h_pred_delta[i]*180)/pi<<endl;
			}
			if(h_a[i+1]>0.8)
			{
				//h_a[i+1]=h_a[i]+c_h;
				Differentialpostfault(&h_pred_delta[i],&h_pred_omega[i],h_pred_delta[i-1],h_pred_omega[i-1],omega0,c_h);
				//cout<<"The predicted value is "<<(h_pred_delta[i]*180)/pi<<endl;
			}

		}
		gettimeofday(&end,NULL);
		fSequential_time[k] += ((end.tv_sec*1e6+end.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		gettimeofday(&start,NULL);
		CHECK(hipMemcpy(d_diff_delta,h_diff_delta,num_steps_bytes_coarse+8,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_diff_omega,h_diff_omega,num_steps_bytes_coarse+8,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_pred_delta,h_pred_delta,num_steps_bytes_coarse+8,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_pred_omega,h_pred_omega,num_steps_bytes_coarse+8,hipMemcpyHostToDevice));
		gettimeofday(&end,NULL);
		double time = 0;
		time = ((end.tv_sec*1e6+end.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		//	int ilen = 256;
		//      dim3 block (ilen,1,1);
		//    dim3 grid ((num_steps+block.x-1)/block.x,1,1);
		CHECK(hipEventRecord(kernel_start));
		gpucorrection<<<grid,block>>>(d_diff_delta,d_diff_omega,d_pred_delta,d_pred_omega,d_corec_delta,d_corec_omega,num_steps);
		CHECK(hipEventRecord(kernel_stop));
		CHECK(hipEventSynchronize(kernel_stop));
		CHECK(hipEventElapsedTime(&fElapsedTime,kernel_start,kernel_stop));
		cout<<"Elapsed time is for correction is " <<fElapsedTime<<endl;
		gettimeofday(&start,NULL);
		CHECK(hipMemcpy(h_corec_delta,d_corec_delta,num_steps_bytes_coarse+8,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_corec_omega,d_corec_omega,num_steps_bytes_coarse+8,hipMemcpyDeviceToHost));
		gettimeofday(&end,NULL);
		time+= ((end.tv_sec*1e6+end.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		cout<<"Correction memory copy time is: "<<time<<" ms"<<endl;
		/*for (int i=0;i<num_steps;i++)
		  {
		  corec_delt[i+1] = h_diff_delta[i]+pred_delt[i];
		  corec_omega[i+1] = h_diff_omega[i]+pred_omega[i];
		//cout<< "The corrected grid values are "<< (corec_delt[i+1]*180)/pi<<" for time"<<h_a[i+1]<<"for array element "<<i<<endl;
		}*/

		fSequential_time[k] += ((end.tv_sec*1e6+end.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		faverage=0;
		for(int i=10;i<110;i++)
		{
			faverage+=et[i];
		}
		fsum[k]=faverage/100;
		cout<<"The gpu execution time is "<<fsum[k]<<"\t"<<"sequential time is "<<fSequential_time[k]<<" ms"<<endl;
		tet[k]=fsum[k]+fSequential_time[k]+fMemoryCopyTime[k];
		cout<<"the elapsed time is "<<tet[k]<<" ms"<<endl;

		CHECK(hipEventDestroy(kernel_start));
		CHECK(hipEventDestroy(kernel_stop));
		CHECK(hipFree(d_omega));
		CHECK(hipFree(d_delta));
		CHECK(hipFree(d_fine_tempd));
		CHECK(hipFree(d_fine_tempo));
		CHECK(hipFree(d_del_fine));
		CHECK(hipFree(d_omega_fine));
		CHECK(hipFree(d_a));
		CHECK(hipFree(d_diff_delta));
		CHECK(hipFree(d_diff_omega));
		CHECK(hipFree(d_pred_delta));
		CHECK(hipFree(d_pred_omega));
		CHECK(hipFree(d_corec_delta));
		CHECK(hipFree(d_corec_omega));
		CHECK(hipDeviceReset());
	}
	/*	delete[] h_omega;
		delete[] h_delta;
		delete[] h_a;
		delete[] h_del_fine;
		delete[] h_omega_fine;
		delete[] h_fine_tempd;
		delete[] h_fine_tempo;
		delete[] h_pred_delta;
		delete[] h_pred_omega;
		delete[] h_corec_delta;
		delete[] h_corec_omega;
		delete[] h_diff_delta;
		delete[] h_diff_omega;*/



}







