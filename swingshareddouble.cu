#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "iostream"
#include "iomanip"
#include "cmath"
#include <stdio.h>
using namespace std;
#define pi 3.14159265358979323846
#define tile 256
#define CHECK(call) \
{                                                                        \
	const hipError_t error = call;                                       \
	if (error != hipSuccess)                                             \
	{                                                                     \
		printf("Error: %s:%d, ", __FILE__, __LINE__);                      \
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
		exit(1);                                                           \
	}                                                                     \
}

// When the fault is occured for GPU function
__device__ void Differentiald(double *deltapresent,double *omegapresent,double deltaprevious,double omegaprevious,double omega0,double c_h)
{
	double temp,temp1,ddeltapresent,domegapresent,ddeltaprevious,domegaprevious;
	ddeltaprevious =omegaprevious-omega0;
	temp=deltaprevious+(c_h)*(ddeltaprevious);
	domegaprevious =((pi*60)/5)* (0.8-0.65*sin(temp));
	temp1 = omegaprevious+(c_h*(domegaprevious));
	ddeltapresent = temp1-omegaprevious;
	*deltapresent = deltaprevious + (c_h/2)*(ddeltaprevious+ddeltapresent);
	domegapresent =((pi*60)/5)* (0.8-(0.65*sin(*deltapresent)));
	*omegapresent = omegaprevious+(c_h/2)*(domegaprevious+domegapresent);
}


//Once the fault is cleared for GPU function

__device__ void Differentialpostfaultd(double *deltapresent,double *omegapresent,double deltaprevious,double omegaprevious,double omega0,double c_h)
{
	double temp,temp1,ddeltapresent,domegapresent,ddeltaprevious,domegaprevious;
	ddeltaprevious =omegaprevious-omega0;
	temp=deltaprevious+(c_h)*(ddeltaprevious);
	domegaprevious =((pi*60)/5)* (0.8-1.4625*sin(temp));
	temp1 = omegaprevious+(c_h*(domegaprevious));
	ddeltapresent = temp1-omegaprevious;
	*deltapresent = deltaprevious + (c_h/2)*(ddeltaprevious+ddeltapresent);
	domegapresent =((pi*60)/5)* (0.8-(1.4625*sin(*deltapresent)));
	*omegapresent = omegaprevious+(c_h/2)*(domegaprevious+domegapresent);
}



__global__ void gpuparareal(double *g_delta,double *g_omega,double *g_a,const double omega0,const double f_h,double *g_fine_tempd,double *g_fine_tempo,double *g_del_fine,double *g_omega_fine,double *g_diff_delta,double *g_diff_omega,int num_steps, const double c_h,double *g_correct_delta)
{
	const int idx = threadIdx.x + (blockIdx.x*blockDim.x);
	const int tid = threadIdx.x;
	if(idx>=num_steps)
	{
		return;
	}
	double tempd,tempo,tint,tfin,fine_step;
	bool flag = (g_a[idx]<0.8);
	__shared__ double s_fine_tempo[tile];
	__shared__ double s_fine_tempd[tile];
	__shared__ double s_predict_delta[tile+2];
	__shared__ double s_predict_omega[tile+2];
	__shared__ double s_a[tile];
	__shared__ double s_correct_delta[tile+2];
	__shared__ double s_correct_omega[tile+2];
	s_a[tid]=g_a[idx];
	tint = g_a[idx];
	int condition = num_steps/tile;
	tfin = g_a[idx+1];
	tempd = g_delta[idx];
	tempo = g_omega[idx];
	for (int tw=0;tw<1;tw++)
	{
//	int tw =1;
		if(flag)
		{
			int umax = round((tfin-tint)/f_h);
			for (int u=0;u<umax;u++)
			{
				fine_step = tint+f_h;
				Differentiald(&s_fine_tempd[tid],&s_fine_tempo[tid],tempd,tempo,omega0,f_h);
				tempd=s_fine_tempd[tid];
				tempo=s_fine_tempo[tid];
				tint=fine_step;
			}
		}

		if(!flag)
		{
			int umax = round((tfin-tint)/f_h);
			for (int u=0;u<umax;u++)
			{
				fine_step = tint+f_h;
				Differentialpostfaultd(&s_fine_tempd[tid],&s_fine_tempo[tid],tempd,tempo,omega0,f_h);
				tempd=s_fine_tempd[tid];
				tempo=s_fine_tempo[tid];
				tint=fine_step;
			}
		}
		g_del_fine[idx+1]=s_fine_tempd[tid];
		g_omega_fine[idx+1]=s_fine_tempo[tid];
		g_diff_delta[idx]=s_fine_tempd[tid] - g_delta[idx+1];
		g_diff_omega[idx]=s_fine_tempo[tid] - g_omega[idx+1];
		//printf("idx = %d The value of fine is %f for time %f for local threadID %d for loop iteration %d condition %d\n",idx,(tempd*180/pi),tfin,tid,tw,condition);
		if(tid == 0 )
		{
			s_predict_delta[0] = s_fine_tempd[0];
			s_predict_omega[0] = s_fine_tempo[0];
		//	printf("The initial delta and omega are %f and %f\n", s_predict_delta[0],s_predict_omega[0]);
			s_a[1]=s_a[0]+c_h;
		//	printf("the value of time step is %f\n",s_a[1]);
			for (int i=1;i<tile;i++)
			{
				s_a[i+1]=s_a[i]+c_h; //a[i] contains all the time step required for coarse grid calculation
				if(s_a[i+1]<=0.8)
				{
					Differentiald(&s_predict_delta[i],&s_predict_omega[i],s_predict_delta[i-1],s_predict_omega[i-1],omega0,c_h);
		//			printf("The predicted value is %f for s_a[i+1] is %f\n",(s_predict_delta[i]*180)/pi,s_a[i+1]);
				}
				if(s_a[i+1]>0.8)
				{
					Differentialpostfaultd(&s_predict_delta[i],&s_predict_omega[i],s_predict_delta[i-1],s_predict_omega[i-1],omega0,c_h);
		//			printf("The predicted value is %f for s_a[i+1] is %f\n",(s_predict_delta[i]*180)/pi,s_a[i+1]);
				}

			}
		}
		__syncthreads();
		s_correct_delta[tid] = s_predict_delta[tid]+g_diff_delta[tid];
		g_correct_delta[idx+1] = s_correct_delta[tid];
		s_correct_omega[tid] = s_predict_omega[tid]+g_diff_omega[tid];		
		//printf("The corrected value is %f for local thread ID %d\n",(s_correct_delta[tid]*180)/pi,tid);
		
	}
}

// When the fault is occured
void Differential(double *deltapresent,double *omegapresent,double deltaprevious,double omegaprevious,double omega0,double c_h)
{
	double temp,temp1,ddeltapresent,domegapresent,ddeltaprevious,domegaprevious;
	ddeltaprevious =omegaprevious-omega0;
	temp=deltaprevious+(c_h)*(ddeltaprevious);
	domegaprevious =((pi*60)/5)* (0.8-0.65*sin(temp));
	temp1 = omegaprevious+(c_h*(domegaprevious));
	ddeltapresent = temp1-omegaprevious;
	*deltapresent = deltaprevious + (c_h/2)*(ddeltaprevious+ddeltapresent);
	domegapresent =((pi*60)/5)* (0.8-(0.65*sin(*deltapresent)));
	*omegapresent = omegaprevious+(c_h/2)*(domegaprevious+domegapresent);
}

//Once the fault is cleared

void Differentialpostfault(double *deltapresent,double *omegapresent,double deltaprevious,double omegaprevious,double omega0,double c_h)
{
	double temp,temp1,ddeltapresent,domegapresent,ddeltaprevious,domegaprevious;
	ddeltaprevious =omegaprevious-omega0;
	temp=deltaprevious+(c_h)*(ddeltaprevious);
	domegaprevious =((pi*60)/5)* (0.8-1.4625*sin(temp));
	temp1 = omegaprevious+(c_h*(domegaprevious));
	ddeltapresent = temp1-omegaprevious;
	*deltapresent = deltaprevious + (c_h/2)*(ddeltaprevious+ddeltapresent);
	domegapresent =((pi*60)/5)* (0.8-(1.4625*sin(*deltapresent)));
	*omegapresent = omegaprevious+(c_h/2)*(domegaprevious+domegapresent);
}


int main()
{
	hipEvent_t kernel_start;
	hipEvent_t kernel_stop;
	struct timeval start,end;
	double tint,tfin,omega0;
	float fElapsedTime,faverage,fsum[10];
	float fMemoryCopyTime[10];
	float fSequential_time[10],tet[10];
	//host variables
	double *h_omega,*h_delta,*h_a,c_h,f_h,dint,*h_del_fine,*h_omega_fine,*h_diff_delta,*h_diff_omega,*h_fine_tempd,*h_fine_tempo,et[110],*h_correct_delta;
	//device variable
	double *d_omega,*d_delta,*d_a,*d_del_fine,*d_omega_fine,*d_fine_tempd,*d_fine_tempo,*d_diff_delta,*d_diff_omega,*d_correct_delta;
	cout<<"The initial time value is : "<<endl;
	cin>>tint;
	cout<<"The final time value is: "<<endl;
	cin>>tfin;
	cout<<"The coarse grid time step value is: "<<endl;
	cin>>c_h;
	cout<<"The fine grid step size value is: "<<endl;
	cin>>f_h;
	cout<<"Enter the intial value of delta in degrees: "<<endl;
	cin>>dint;
	int num_steps = ((tfin-tint)/c_h)+1;
	cout<<"the number of steps for coarse : "<<num_steps<<endl;
	size_t num_steps_bytes_coarse = num_steps*sizeof(double);
	int fine_size = ((tfin-tint)/f_h)+1;
	cout<<"The number of steps for fine : "<<fine_size<<endl;
	size_t num_steps_bytes_fine = fine_size*sizeof(double);
	h_omega = new double[num_steps];
	h_delta = new double[num_steps];
	h_a = new double[num_steps];
	h_del_fine = new double[num_steps];
	h_omega_fine = new double[num_steps];
	h_fine_tempd= new double[fine_size];
	h_fine_tempo=new double[fine_size];
	h_diff_delta = new double[num_steps];
	h_diff_omega = new double [num_steps];
	h_correct_delta = new double [num_steps];
	omega0=2*pi*60;
	h_omega[0]=omega0;
	h_delta[0]=(dint*pi)/180;
	cout<<"The value in radians is: "<<h_delta[0]<<endl;
	h_a[0] =0;
	h_a[0]=tint;
	num_steps =num_steps - 1;
	fine_size =fine_size - 1;
	cout<<num_steps<<endl;
	for(int k=0;k<2;k++)
	{
		fMemoryCopyTime[k]=0;
		fSequential_time[k]=0;
		gettimeofday(&start,NULL);
		for (int i=0;i<num_steps;i++)
		{
			h_a[i+1]=h_a[i]+c_h; //a[i] contains all the time step required for coarse grid calculation
			if(h_a[i+1]<=0.8)
			{
				//cout << "a= " <<h_a[i+1]<<__LINE__<<endl;
				//h_a[i+1]=h_a[i]+c_h; //a[i] contains all the time step required for coarse grid calculation
				Differential(&h_delta[i+1],&h_omega[i+1],h_delta[i],h_omega[i],omega0,c_h);
				//cout<< "The coarse grid values are "<< (h_delta[i+1]*180)/pi<<" for time"<<h_a[i+1]<<"for array element "<<i<<"for k value "<<k<<endl;
				//cout<<"break 2"<<endl;

			}
			if(h_a[i+1]>0.8)
			{
				//cout << "a= " <<h_a[i]<<__LINE__<<endl;
				//h_a[i+1]=h_a[i]+c_h;
				Differentialpostfault(&h_delta[i+1],&h_omega[i+1],h_delta[i],h_omega[i],omega0,c_h);
				//cout<< "The coarse grid values are "<< (h_delta[i+1]*180)/pi<<" for time"<<h_a[i+1]<<"for array element "<<i<<"for k value "<<k<<endl; 
			}

		}
		gettimeofday(&end,NULL);
		fSequential_time[k] = ((end.tv_sec*1e6+end.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		cout<<" The Sequential Execution time is : "<<fSequential_time[k]<<" ms"<<endl;
		CHECK(hipEventCreate(&kernel_start));
		CHECK(hipEventCreate(&kernel_stop));
		//Allocating memory on GPU for device variables

		CHECK(hipMalloc((double**)&d_delta,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_omega,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_fine_tempd,num_steps_bytes_fine+8));
		CHECK(hipMalloc((double**)&d_fine_tempo,num_steps_bytes_fine+8));
		CHECK(hipMalloc((double**)&d_del_fine,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_omega_fine,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_a,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_diff_delta,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_diff_omega,num_steps_bytes_coarse+8));
		CHECK(hipMalloc((double**)&d_correct_delta,num_steps_bytes_coarse+8));
		//copying the data to device from host
		gettimeofday(&start,NULL);
		CHECK(hipMemcpy(d_delta,h_delta,num_steps_bytes_coarse+8,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_omega,h_omega,num_steps_bytes_coarse+8,hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_a,h_a,num_steps_bytes_coarse+8,hipMemcpyHostToDevice));
		gettimeofday(&end,NULL);
		fMemoryCopyTime[k] = ((end.tv_sec*1e6+end.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		//Kernel call
		//Kernel call
		int ilen = 128;
		dim3 block (ilen,1,1);
		dim3 grid ((num_steps+block.x-1)/block.x,1,1);
		cout << "1D Grid Dimension" << endl;
		cout << "\tNumber of Blocks along X dimension: " << grid.x << endl;
		cout << "1D Block Dimension" << endl;
		cout << "\tNumber of threads along X dimension: " << block.x << endl;
		//kernel function
		et[0]=0;
		for(int i=0;i<110;i++)
		{
			CHECK(hipEventRecord(kernel_start));
			gpuparareal<<<grid,block>>>(d_delta,d_omega,d_a,omega0,f_h,d_fine_tempd,d_fine_tempo,d_del_fine,d_omega_fine,d_diff_delta,d_diff_omega,num_steps,c_h,d_correct_delta);
			CHECK(hipEventRecord(kernel_stop));
			CHECK(hipEventSynchronize(kernel_stop));
			CHECK(hipEventElapsedTime(&fElapsedTime,kernel_start,kernel_stop));
			et[i]=fElapsedTime;
		}
		CHECK(hipMemcpy(h_fine_tempd,d_fine_tempd,num_steps_bytes_fine+8,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_fine_tempo,d_fine_tempo,num_steps_bytes_fine+8,hipMemcpyDeviceToHost));
//		gettimeofday(&start,NULL);
		CHECK(hipMemcpy(h_diff_delta,d_diff_delta,num_steps_bytes_coarse+8,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_diff_omega,d_diff_omega,num_steps_bytes_coarse+8,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_del_fine,d_del_fine,num_steps_bytes_coarse+8,hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(h_omega_fine,d_omega_fine,num_steps_bytes_coarse+8,hipMemcpyDeviceToHost));
		gettimeofday(&start,NULL);
		CHECK(hipMemcpy(h_correct_delta,d_correct_delta,num_steps_bytes_coarse+8,hipMemcpyDeviceToHost));
		//CHECK(cudaMemcpy(h_correct_omega,d_correct_omega,num_steps_bytes_coarse=4,cudaMemcpyDeviceToHost));
		gettimeofday(&end,NULL);
	/*	for(int i=1;i<=num_steps;i++)
		{
	//	cout<<(h_correct_delta[i]*180)/pi<<endl;
		}*/
		fMemoryCopyTime[k]+= ((end.tv_sec*1e6+end.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		cout<< "Memory transfer time =  " << fMemoryCopyTime[k] <<" ms"<<endl;
		fSequential_time[k] += ((end.tv_sec*1e6+end.tv_usec)-(start.tv_sec*1e6+start.tv_usec))/1000;
		faverage=0;
		for(int i=10;i<110;i++)
		{
			faverage+=et[i];
		}
		fsum[k]=faverage/100;
		cout<<"The gpu execution time is "<<fsum[k]<<"\t"<<"sequential time is "<<fSequential_time[k]<<" ms"<<endl;
		tet[k]=fsum[k]+fSequential_time[k]+fMemoryCopyTime[k];
		cout<<"the elapsed time is "<<tet[k]<<" ms"<<endl;

		CHECK(hipEventDestroy(kernel_start));
		CHECK(hipEventDestroy(kernel_stop));
		CHECK(hipFree(d_omega));
		CHECK(hipFree(d_delta));
		CHECK(hipFree(d_fine_tempd));
		CHECK(hipFree(d_fine_tempo));
		CHECK(hipFree(d_del_fine));
		CHECK(hipFree(d_omega_fine));
		CHECK(hipFree(d_a));
		CHECK(hipFree(d_diff_delta));
		CHECK(hipFree(d_diff_omega));
		CHECK(hipDeviceReset());
	}
/*	delete[] h_omega;
	delete[] h_delta;
	delete[] h_a;
	delete[] h_del_fine;
	delete[] h_omega_fine;
	delete[] h_fine_tempd;
	delete[] h_fine_tempo;
	delete[] h_diff_delta;
	delete[] h_diff_omega;
*/
}























